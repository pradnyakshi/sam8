//%%cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#include <cstdlib>
#include <chrono>
#include <omp.h>
using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

void addS(int* A, int* B, int* C, int size) {
    for(int i=0;i<size;i++){
    	C[i] = A[i] + B[i];
    }

}


void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 1000;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 10000;
    int* A, * B, * C, *D;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];
    D = new int[vectorSize];
    
    initialize(A, vectorSize);
    initialize(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);
    
    auto startSeq = chrono::steady_clock::now();
    // sequential 
    addS(A,B,D,N);
    auto endSeq = chrono::steady_clock::now();
    chrono::duration<double, micro> fp = endSeq - startSeq;
    cout << "Sequential Time: " << fp.count() << " microseconds" << endl;
    

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);
    
    
    auto startParallel = chrono::steady_clock::now();

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
    
    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);
    
    auto endParallel = chrono::steady_clock::now();
    chrono::duration<double, micro> fp1 = endParallel - startParallel;
    cout << "Parallel Time: " << fp1.count() << " microseconds" << endl;
    // cout << "Addition: ";
    // print(C, N);

    for(int i =0;i<N;i++){
        if(C[i]!=D[i]){
            cout<<"Wrong at "<<i<<endl;
        }
    }
    
    double SortSpeedup = fp.count() / fp1.count();
    cout << "Speedup : " << SortSpeedup << endl;

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] D;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}